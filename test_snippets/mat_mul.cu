
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <chrono>
#include <cstdlib>

// CUDA kernel for matrix multiplication
__global__ void matmul(const int N, const float* A, const float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

// CPU implementation for matrix multiplication
void matmul_cpu(const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C, int N) {
    for (int row = 0; row < N; ++row) {
        for (int col = 0; col < N; ++col) {
            float sum = 0.0f;
            for (int k = 0; k < N; ++k) {
                sum += A[row * N + k] * B[k * N + col];
            }
            C[row * N + col] = sum;
        }
    }
}

int main(int argc, char* argv[]) {
    const int N = (argc > 1) ? atoi(argv[1]) : 1024;  // Set default size if not provided
    std::vector<float> A(N * N); // Initialize matrix A
    std::vector<float> B(N * N); // Initialize matrix B
    std::vector<float> C(N * N, 0.0f); // Matrix to store the result

    // Initialize matrices
    for (int i = 0; i < N * N; ++i) {
        A[i] = rand() % 100;
        B[i] = rand() % 200;
    }

    // CPU Timing
//    auto start_cpu = std::chrono::high_resolution_clock::now();
//    matmul_cpu(A, B, C, N);
//    auto end_cpu = std::chrono::high_resolution_clock::now();
//    std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;

//    std::cout << "CPU matrix multiplication took: " << cpu_duration.count() << " seconds" << std::endl;
//    std::cout << "FIRST ELEMENT: " << C[0] << std::endl;
//    std::cout << "MIDDLE ELEMENT: " << C[N*N/2] << std::endl;
//    std::cout << "LAST ELEMENT: " << C[N*N-1] << std::endl;

    // Device pointers
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void**)&d_A, sizeof(float) * N * N);
    hipMalloc((void**)&d_B, sizeof(float) * N * N);
    hipMalloc((void**)&d_C, sizeof(float) * N * N);

    // Copy data to device
    hipMemcpy(d_A, A.data(), sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), sizeof(float) * N * N, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y); // Grid size

    // GPU Timing
    auto start_gpu = std::chrono::high_resolution_clock::now();
    // Launch the kernel
    matmul<<<gridSize, blockSize>>>(N, d_A, d_B, d_C);
    // Wait for the GPU to finish
    hipDeviceSynchronize();
    auto end_gpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> gpu_duration = end_gpu - start_gpu;

    // Copy the result back to host
    hipMemcpy(C.data(), d_C, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    std::cout << "GPU matrix multiplication took: " << gpu_duration.count() << " seconds" << std::endl;
    std::cout << "FIRST ELEMENT: " << C[0] << std::endl;
    std::cout << "MIDDLE ELEMENT: " << C[N*N/2] << std::endl;
    std::cout << "LAST ELEMENT: " << C[N*N-1] << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

